#include "hip/hip_runtime.h"
#include<iostream>
#include <stdio.h>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <errno.h>
#include <string.h>
#include <unistd.h>
#include <iomanip>

#include "include/params.cuh"
#include "include/DataStructure.cuh"
#include "include/MCMPC.cuh"
#include "include/init.cuh"
#include "include/cuSolverForMCMPC.cuh"

#define Linear

int main(int argc, char **argv)
{
    /*データ書き込みファイルの定義*/
    FILE *fp;
    time_t timeValue;
    struct tm *timeObject;
    time( &timeValue );
    timeObject = localtime( &timeValue );
    char filename1[35];
    sprintf(filename1,"data_system_%d%d_%d%d.txt",timeObject->tm_mon + 1, timeObject->tm_mday, timeObject->tm_hour,timeObject->tm_min);
    fp = fopen(filename1,"w");


    float params[dim_param], state[dim_state], /*h_constraint[NUM_CONST],*/ h_matrix[dim_weight_matrix];
    Mat_sys_A( params );
    init_state( state );
    // init_constraint( h_constraint );
    init_Weight_matrix( h_matrix );
    hipMemcpyToSymbol(HIP_SYMBOL(d_param), &params, dim_param * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_matrix), h_matrix, dim_weight_matrix * sizeof(float));

#ifdef Linear
    float opt[HORIZON], Error[HORIZON];
    init_opt( opt );
#endif


    /* GPUの設定 */
    unsigned int numBlocks, randomBlocks, randomNums/*, minId_cpu*/;
    int Blocks;
    randomNums = N_OF_SAMPLES * dim_U * HORIZON;
    randomBlocks = countBlocks(randomNums, THREAD_PER_BLOCKS);
    numBlocks = countBlocks(N_OF_SAMPLES, THREAD_PER_BLOCKS);
    printf("#NumBlocks = %d\n", numBlocks);
    Blocks = numBlocks;

    /* CPU to GPU dataExchanger */
    Data1 *h_dataFromBlocks;
    Data1 *d_dataFromBlocks;

    h_dataFromBlocks = (Data1 *)malloc(sizeof(Data1)*numBlocks);
    hipMalloc(&d_dataFromBlocks, sizeof(Data1) * numBlocks);

    /* hiprand の設定 */
    hiprandState *devStates;
    hipMalloc((void **)&devStates, randomNums * sizeof(hiprandState));
    setup_kernel<<<randomBlocks, THREAD_PER_BLOCKS>>>(devStates,rand());
    hipDeviceSynchronize();

    /* Covariance の定義 */
    float *h_hat_Q, *Diag_D;
    float *device_cov;
    float *device_diag_eig = NULL;
    h_hat_Q = (float *)malloc(sizeof(float)*dim_hat_Q);
    Diag_D = (float *)malloc(sizeof(float)*dim_hat_Q);
    hipMalloc(&device_cov, sizeof(float)*dim_hat_Q);
    hipMalloc(&device_diag_eig, sizeof(float)*dim_hat_Q);
    /*hipMalloc(&d_hat_Q, sizeof(float)*dim_hat_Q);*/

    setup_init_Covariance<<<HORIZON, HORIZON>>>(d_hat_Q);

    /* 準最適制御入力列 */
    float *Us_host, *Us_device;
    Us_host = (float *)malloc(sizeof(float) * HORIZON);
    for(int i = 0; i < HORIZON; i++){
        Us_host[i] = 0.0f;
    }
    hipMalloc(&Us_device, sizeof(float) * HORIZON);


    float var;
    float now_u;
    for(int i = 0; i < Blocks; i++){
        for(int k = 0; k < HORIZON; k++){
            h_dataFromBlocks[i].Input[k] = 0.0f;
        }
    }

    /* 固有値の取得 */
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;

    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    const int m = HORIZON;
    const int lda = m;

    float eig_vec[m] = { };

    float *d_A = NULL;
    float *d_W = NULL;
    int *devInfo = NULL;
    float *d_work = NULL;
    int lwork = 0;

    int info_gpu = 0;

    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(float) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_W, sizeof(float) * m);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    for(int time = 0; time < TIME; time++){
        for(int repeat = 0; repeat < Recalc; repeat++){
            var = Variavility;
            hipMemcpy(d_dataFromBlocks, h_dataFromBlocks, sizeof(Data1)*numBlocks, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            // MCMPC_GPU<<<numBlocks, THREAD_PER_BLOCKS>>>(state, devStates, d_dataFromBlocks, var, Blocks, d_hat_Q);
            MCMPC_GPU_Linear_Example<<<numBlocks, THREAD_PER_BLOCKS>>>(state, devStates, d_dataFromBlocks, var, Blocks, d_hat_Q);
            hipDeviceSynchronize();
            hipMemcpy(h_dataFromBlocks, d_dataFromBlocks, sizeof(Data1) * numBlocks, hipMemcpyDeviceToHost);
            weighted_mean(h_dataFromBlocks, Blocks, Us_host);
            hipMemcpy(Us_device, Us_host, sizeof(float) * HORIZON, hipMemcpyHostToDevice);
            calc_Var_Cov_matrix<<<HORIZON, HORIZON>>>(device_cov, d_dataFromBlocks, Us_device, Blocks);
            hipDeviceSynchronize();
            hipMemcpy(h_hat_Q, device_cov, sizeof(float)*dim_hat_Q, hipMemcpyDeviceToHost);
            // get_eigen_values(h_hat_Q, Diag_D);
            /* 固有値の取得 */
            /*hipsolverHandle_t cusolverH = NULL;
            hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
            hipError_t cudaStat1 = hipSuccess;
            hipError_t cudaStat2 = hipSuccess;
            hipError_t cudaStat3 = hipSuccess;
            const int m = HORIZON;
            const int lda = m;

            float eig_vec[m] = { };

            float *d_A = NULL;
            float *d_W = NULL;
            int *devInfo = NULL;
            float *d_work = NULL;
            int lwork = 0;

            int info_gpu = 0;

            /*cusolver_status = hipsolverDnCreate(&cusolverH);
            assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);*/

            cudaStat1 = hipMemcpy(d_A, h_hat_Q, sizeof(float) * lda * m, hipMemcpyHostToDevice);
            assert(hipSuccess == cudaStat1);

            hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
            hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

            cusolver_status = hipsolverDnSsyevd_bufferSize(
                cusolverH,
                jobz,
                uplo,
                m,
                d_A,
                lda,
                d_W,
                &lwork);
            assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

            cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
            assert(hipSuccess == cudaStat1);

            cusolver_status = hipsolverDnSsyevd(
                cusolverH,
                jobz,
                uplo,
                m,
                d_A,
                lda,
                d_W,
                d_work,
                lwork,
                devInfo);

            cudaStat1 = hipDeviceSynchronize();
            assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
            assert(hipSuccess == cudaStat1);

            cudaStat1 = hipMemcpy(eig_vec, d_W, sizeof(double)*m, hipMemcpyDeviceToHost);
            cudaStat2 = hipMemcpy(Diag_D, d_A, sizeof(double)*lda*m, hipMemcpyDeviceToHost);
            cudaStat3 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
            assert(hipSuccess == cudaStat1);
            assert(hipSuccess == cudaStat2);
            assert(hipSuccess == cudaStat3);
            make_Diagonalization<<<HORIZON,HORIZON>>>(d_W, d_A);
            hipMemcpy(h_hat_Q, d_A, sizeof(float)*lda*m, hipMemcpyDeviceToHost);

            hipMemcpy(device_diag_eig, h_hat_Q, sizeof(float)*dim_hat_Q, hipMemcpyHostToDevice);
            hipMemcpy(device_cov, Diag_D, sizeof(float)*dim_hat_Q, hipMemcpyHostToDevice);
            pwr_matrix_answerB<<<HORIZON,HORIZON>>>(device_cov, device_diag_eig);
            hipDeviceSynchronize();
            pwr_matrix_answerA<<<HORIZON,HORIZON>>>(device_diag_eig, device_cov);
            hipDeviceSynchronize();
            hipMemcpy(d_hat_Q, device_diag_eig, sizeof(float)*dim_hat_Q, hipMemcpyDeviceToHost);

            fprintf(fp,"%f %f %f %f %f %f %f %f %f %f\n",Us_host[0], Us_host[1],
                    Us_host[2], Us_host[3], Us_host[4], Us_host[5], Us_host[6], Us_host[7], Us_host[8], Us_host[9]);

#ifdef Linear
            float RSME;
            for(int d = 0; d < HORIZON; d++){
                Error[d] = Us_host[d] - opt[d];
                RSME += powf(Error[d],2);
            }
            printf("RSME == %f\n", RSME / HORIZON);
#endif
        }
        now_u = Us_host[0];
        calc_Linear_example(state, now_u, params, state);
        for(int i = 0; i < Blocks; i++){
            for(int k = 0; k < HORIZON - 1; k++){
                h_dataFromBlocks[i].Input[k] = Us_host[k+1];
            }
            h_dataFromBlocks[i].Input[HORIZON-1] = Us_host[HORIZON - 1];
        }
    }
    if (d_A    ) hipFree(d_A);
    if (d_W    ) hipFree(d_W);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    fclose(fp);
    // fclose(hp);
    hipDeviceReset();
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    return 0;
}