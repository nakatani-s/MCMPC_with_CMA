#include "hip/hip_runtime.h"
/*
include "MCMPC.cuh"
*/
#include<stdio.h>
#include "../include/MCMPC.cuh" 

__global__ void setup_kernel(hiprandState *state,int seed) {
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence number,
     no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

unsigned int countBlocks(unsigned int a, unsigned int b) {
	unsigned int num;
	num = a / b;
	if (a < b || a % b > 0)
		num++;
	return num;
}

void weighted_mean(Data1 *h_Data, int Blocks, float *Us_host)
{
    float total_weight = 0.0f;
    float temp[HORIZON] = {};
    for(int i = 0; i < Blocks; i++){
        if(isnan(h_Data[i].W))
        {
            total_weight += 0.0f;
        }else{
            total_weight += h_Data[i].W;
        }
    }

    for(int i = 0; i < HORIZON; i++)
    {
        for(int k = 0; k < Blocks; k++)
        {
            if(isnan(h_Data[k].W))
            {
                temp[i] += 0.0f;
            }else{
                temp[i] += h_Data[k].W * h_Data[k].Input[i] / total_weight;
            }
            Us_host[i] = temp[i]; 
        }
    }
}

__device__ float generate_u(int t, float mean, float var, float *d_cov, float *z)
{
    int count_index;
    count_index = t * HORIZON;
    float ret, sec_term;
    for(int k = 0; k < HORIZON; k++)
    {
        sec_term += d_cov[count_index + k]*z[k];
    }
    ret = mean + var * sec_term;
    return ret;
}

__device__ float gen_u(unsigned int id, hiprandState *state, float ave, float vr) {
    float u;
    hiprandState localState = state[id];
    u = hiprand_normal(&localState) * vr + ave;
    return u;
}

__global__ void setup_init_Covariance(float *Mat)
{
    unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
    //float values;
    /*if(threadIdx.x == 0 && blockIdx.x ==0)
    {
        values[threadIdx.x] = 1.0f;
    }*/
    if(threadIdx.x == blockIdx.x)
    {
        Mat[id] = 1.0f;
        values[threadIdx.x] = 1.0f;
    }else{
        Mat[id] = 0.0f;
        values[threadIdx.x] = 0.0f;
    }
    __syncthreads();
    /*if(threadIdx.x == 0)
    {
       for(int i =0; i < blockDim.x; i++)
           Mat[id] = values[i];
    }  */   
    
}

__global__ void MCMPC_GPU_Linear_Example(float x, float y, float w, hiprandState *devs, Data1 *d_Datas, float var, int Blocks, float *d_cov, float *d_param, float *d_matrix)
{
    unsigned int id = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int seq;
    seq = id;
    float qx = 0.0f;
    float total_cost = 0.0f;
    float u[HORIZON]= { };
    float block_var;
    // int Powers;
    //printf("hoge id=%d\n", id);
    float d_state_here[dim_state] = {x,y,w};
    float get_state[dim_state] = {};
    float z[HORIZON] = { };

    for(int t = 0; t < HORIZON; t++)
    {
        block_var = var;
        for(int t_x = 0; t_x < HORIZON; t_x++)
        {
            z[t_x] = gen_u(seq, devs, 0, 1.0f);
            seq += 3;
        }
        u[t] = generate_u(t, d_Datas[0].Input[t], var, d_cov, z);
        if(u[t]<-4.0f){
           u[t] = -4.0f;
        }
        if(u[t] > 4.0f){
           u[t] = 4.0f;
        }
        //printf("hoge id=%d @ %f %f\n", id, u[t], z[t]);
        calc_Linear_example(d_state_here, u[t], d_param, get_state);
        //printf("hoge id=%d @ %f %f %f\n", id, u[t], d_param[0], get_state[1]);
        qx += d_matrix[0] * get_state[0] * get_state[0];
        qx += d_matrix[1] * get_state[0] * get_state[1];
        qx += d_matrix[3] * get_state[0] * get_state[1];
        qx += d_matrix[4] * get_state[1] * get_state[1];
        for(int h = 0; h < dim_state; h++){
           d_state_here[h] = get_state[h];
        }
        
        total_cost += qx;

        qx = 0.0f;
    }

    float KL_COST, S, lambda;
    lambda = HORIZON * dim_state;
    S = total_cost / lambda;
    KL_COST = exp(-S);
    W_comp[threadIdx.x] = KL_COST;
    L_comp[threadIdx.x] = total_cost;
    __syncthreads();
    if(threadIdx.x == 0)
    {
        best_thread_id_this_block = 0;
        for(int y = 1; y < blockDim.x; y++){
            if(L_comp[y] < L_comp[best_thread_id_this_block])
            {
                best_thread_id_this_block = y;
            }
        }
    }
    __syncthreads();
    if(threadIdx.x == best_thread_id_this_block)
    {
        Data1 block_best;
        block_best.L = L_comp[best_thread_id_this_block];
        block_best.W = W_comp[best_thread_id_this_block];
        for(int z = 0; z < HORIZON; z++)
        {
            block_best.Input[z] = u[z];
        }
        d_Datas[blockIdx.x] = block_best;

    } 
}
